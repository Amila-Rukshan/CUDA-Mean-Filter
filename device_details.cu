#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

int main()
{
  hipError_t error;
  hipDeviceProp_t dev;
  int dev_cnt = 0;

hipProfilerStart();
  // return device numbers with compute capability >= 1.0
  error = hipGetDeviceCount (&dev_cnt);
  if(error != hipSuccess)
  {
    printf("Error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
  printf("Number of devices: %d\n",dev_cnt);

  // Get properties of each device
  for(int i = 0; i < dev_cnt; i++)
  {
     error = hipGetDeviceProperties(&dev, i);
     if(error != hipSuccess)
     {
        printf("Error: %s\n", hipGetErrorString(error));
        exit(-1);
     }
     printf("\nDevice %d:\n", i);
     printf("name: %s\n",dev.name);
     printf("Compute capability %d.%d\n",dev.major, dev.minor);
     printf("total global memory(KB): %ld\n", dev.totalGlobalMem/1024);
     printf("shared mem per block: %d\n",dev.sharedMemPerBlock);
     printf("regs per block: %d\n", dev.regsPerBlock);
     printf("warp size: %d\n", dev.warpSize);
     printf("max threads per block: %d\n",dev.maxThreadsPerBlock);
     printf("max thread dim z:%d y:%d x:%d\n", dev.maxThreadsDim[0], dev.maxThreadsDim[1], dev.maxThreadsDim[2]);
     printf("max grid size z:%d y:%d x:%d\n", dev.maxGridSize[0],dev.maxGridSize[1], dev.maxGridSize[2]);
     printf("clock rate(KHz):\n",dev.clockRate);
     printf("total constant memory (bytes): %ld\n",dev.totalConstMem);
     printf("multiprocessor count %d\n",dev.multiProcessorCount);
     printf("memory bus width: %d\n",dev.memoryBusWidth);
     printf("memory clock rate (KHz): %d\n",dev.memoryClockRate);
     printf("L2 cache size (bytes): %d\n", dev.l2CacheSize);
     printf("max threads per SM: %d\n", dev.maxThreadsPerMultiProcessor);
  }

hipProfilerStop();
  return 0;

}