#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <hip/hip_runtime.h>

int *a, *b;  // host data
int *c, *c2;  // results

__global__ void vecAdd(int *A,int *B,int *C,int N)
{
   int i = blockDim.x * blockIdx.x + threadIdx.x;
   if(i < N)
      C[i] = A[i] + B[i];
}

void vecAdd_h(int *A1,int *B1, int *C1, int N)
{
   for(int i=0;i<N;i++)
      C1[i] = A1[i] * B1[i];
}

int main(int argc,char **argv)
{
   printf("Begin \n");
   int n=100000000;
   int nBytes = n*sizeof(int);
   int block_size, block_no;
   a = (int *)malloc(nBytes);
   b = (int *)malloc(nBytes);
   c = (int *)malloc(nBytes);
   c2 = (int *)malloc(nBytes);
   int *a_d,*b_d,*c_d;
   block_size=4000;
   block_no = n/block_size;
   dim3 dimBlock(block_size,1,1);
   dim3 dimGrid(block_no,1,1);

   for(int i = 0; i < n; i++ ) {
        a[i] = sin(i)*sin(i);
        b[i] = cos(i)*cos(i);
   }
 printf("Allocating device memory on host..\n");
   hipMalloc((void **)&a_d,n*sizeof(int));
   hipMalloc((void **)&b_d,n*sizeof(int));
   hipMalloc((void **)&c_d,n*sizeof(int));
   printf("Copying to device..\n");
   hipMemcpy(a_d,a,n*sizeof(int),hipMemcpyHostToDevice);
   hipMemcpy(b_d,b,n*sizeof(int),hipMemcpyHostToDevice);
   clock_t start_d=clock();
   printf("Doing GPU Vector add\n");
   vecAdd<<<block_no,block_size>>>(a_d,b_d,c_d,n);
   hipDeviceSynchronize();
   clock_t end_d = clock();
   clock_t start_h = clock();
   printf("Doing CPU Vector add\n");
   vecAdd_h(a,b,c2,n);
   clock_t end_h = clock();
   double time_d = (double)(end_d-start_d)/CLOCKS_PER_SEC;
   double time_h = (double)(end_h-start_h)/CLOCKS_PER_SEC;
   hipMemcpy(c,c_d,n*sizeof(int),hipMemcpyDeviceToHost);
   printf("Number of elements: %d GPU Time: %f CPU Time: %f\n",n,time_d,time_h);
   hipFree(a_d);
   hipFree(b_d);
   hipFree(c_d);
   return 0;
}